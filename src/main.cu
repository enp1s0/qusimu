#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <hip/hip_cooperative_groups.h>
#include <cutf/memory.hpp>

// CUDAの組み込み関数はconstexprではないので
constexpr float sqrt2 = 1.41421356237f;

// 命令は固定長
using inst_t = uint64_t;
using inst_type_t = uint64_t;
using qubit_t = float;
// unary命令
// |63        61|57    30|29       0|
// |  命令種別  | 未使用 | 計算対象 |
// binary命令
// |63        61|60    35|34          30|29       0|
// |  命令種別  | 未使用 | コントロール | 計算対象 |
// ternary命令
// |63        61|60    40|39          35|34          30|29       0|
// |  命令種別  | 未使用 | コントロール | コントロール | 計算対象 |

// 命令種別
constexpr inst_type_t inst_type_nil = 0x0;
constexpr inst_type_t inst_type_x   = 0x1;
constexpr inst_type_t inst_type_z   = 0x2;
constexpr inst_type_t inst_type_h   = 0x3;
constexpr inst_type_t inst_type_cx  = 0x4;
constexpr inst_type_t inst_type_cz  = 0x5;
constexpr inst_type_t inst_type_ccx = 0x6;

__device__ void convert_x(qubit_t* const qubits, const inst_t inst, const std::size_t tid, const cooperative_groups::coalesced_group &all_threads_group){
	// 交換部分の解析
	constexpr auto mask = (~(static_cast<inst_t>(1)<<31));
	const auto xor_mask = inst & mask;

	// TODO : 書き込みと読み込みのどちらで結合アクセスを使うか
	// TODO : 実は処理が「交換」なので，並列数は半分で構わない
	const auto tmp = qubits[tid];
	all_threads_group.sync();
	qubits[tid ^ xor_mask] = tmp;
}
__device__ void convert_z(qubit_t* const qubits, const inst_t inst, const std::size_t tid){
	constexpr auto mask = (~(static_cast<inst_t>(1)<<31));
	const auto target_bits = inst & mask;

	if((tid & target_bits) != 0){
		// TODO : 先頭ビット反転とどちらが速いか
		qubits[tid] = -qubits[tid];
	}
}
__device__ void convert_h(qubit_t* const qubits, const inst_t inst, const std::size_t tid, const cooperative_groups::coalesced_group &all_threads_group){
	// 交換部分の解析
	constexpr auto mask = (~(static_cast<inst_t>(1)<<31));
	const auto target_bits = inst & mask;

	// TODO : 書き込みと読み込みのどちらで結合アクセスを使うか
	// TODO : 実は処理が「交換」なので，並列数は半分で構わない
	const auto p0 = qubits[tid];
	const auto p1 = qubits[tid ^ target_bits];
	all_threads_group.sync();
	if((tid & target_bits) != 0){
		qubits[tid] = (p0 + p1) / sqrt2;
	}else{
		qubits[tid] = (p0 - p1) / sqrt2;
	}
}
__device__ void convert_cx(qubit_t* const qubits, const inst_t inst, const std::size_t tid, const cooperative_groups::coalesced_group &all_threads_group){
	constexpr auto mask = (~(static_cast<inst_t>(1)<<31));
	const auto target_bits = inst & mask;
	// 31bit目から5bitがcontrolなので
	const auto ctrl_bits = static_cast<inst_t>(1) << ((inst >> 30) & 0x1f);

	if(tid & ctrl_bits == 0){
		return;
	}
	const auto p = qubits[tid ^ target_bits];
	all_threads_group.sync();
	qubits[tid] = p;
}
__device__ void convert_cz(qubit_t* const qubits, const inst_t inst, const std::size_t tid){
	constexpr auto mask = (~(static_cast<inst_t>(1)<<31));
	const auto target_bits = inst & mask;
	// 31bit目から5bitがcontrolなので
	const auto ctrl_bits = static_cast<inst_t>(1) << ((inst >> 30) & 0x1f);

	if(tid & ctrl_bits == 0 || tid & target_bits == 0){
		return;
	}
	qubits[tid] = -qubits[tid];
}
__device__ void convert_ccx(qubit_t* const qubits, const inst_t inst, const std::size_t tid, const cooperative_groups::coalesced_group &all_threads_group){
	constexpr auto mask = (~(static_cast<inst_t>(1)<<31));
	const auto target_bits = inst & mask;
	// 31bit目から5bitがcontrolなので
	const auto ctrl_bits_0 = static_cast<inst_t>(1) << ((inst >> 30) & 0x1f);
	const auto ctrl_bits_1 = static_cast<inst_t>(1) << ((inst >> 35) & 0x1f);

	if(tid & ctrl_bits_0 == 0 || tid & ctrl_bits_1 == 0){
		return;
	}
	const auto p = qubits[tid ^ target_bits];
	all_threads_group.sync();
	qubits[tid] = p;
}

__global__ void qusimu_kernel(qubit_t* const qubits, const inst_t* const insts, const std::size_t num_insts, const std::size_t N){
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= N){
		return;
	}
	// 全スレッドでgroupを作る
	const auto all_threads_group = cooperative_groups::coalesced_threads();
	// 命令実行ループ
	for(std::size_t inst_index = 0; inst_index < num_insts;){
		all_threads_group.sync();
		// デコード
		// 全スレッドが同じアドレスへアクセスするためキャッシュをうまく使いましょう
		const auto inst = __ldg(insts + inst_index);
		// |63   61|が命令種別なのでマジックナンバー61
		const auto inst_type = static_cast<inst_type_t>(inst >> 61);

		// X
		if(inst_type == inst_type_x){
			convert_x(qubits, inst, tid, all_threads_group);
			continue;
		}

		// Z
		if(inst_type == inst_type_z){
			convert_z(qubits, inst, tid);
			continue;
		}

		// H
		if(inst_type == inst_type_h){
			convert_h(qubits, inst, tid, all_threads_group);
			continue;
		}

		// CX
		if(inst_type == inst_type_cx){
			convert_cx(qubits, inst, tid, all_threads_group);
			continue;
		}

		// CZ
		if(inst_type == inst_type_cz){
			convert_cz(qubits, inst, tid);
			continue;
		}

		// CCX
		if(inst_type == inst_type_ccx){
			convert_ccx(qubits, inst, tid, all_threads_group);
			continue;
		}
	}
}

int main(){
	std::size_t n, k;
	std::cin >> n >> k;

	// 量子ビットの組み合わせ総数
	const std::size_t N = 1 << n;

	// 量子ビット on デバイスメモリ
	auto d_qubits_uptr = cutf::cuda::memory::get_device_unique_ptr<qubit_t>(N);

	// 発行命令列
	std::vector<inst_t> insts_vec;

	// 読み取り
	for(std::size_t k_index = 0; k_index < k; k_index++){
		char gate[4];
		// 命令種別読み取り
		std::scanf("%s", gate);

		// 解析
		if(gate[0] == 'X' && gate[1] == '\0'){
			std::size_t target;
			std::scanf("%lu", &target);
			insts_vec.push_back(inst_type_x<<61 | (static_cast<inst_t>(1)<<target));
		}else if(gate[0] == 'Z' && gate[1] == '\0'){
			std::size_t target;
			std::scanf("%lu", &target);
			insts_vec.push_back(inst_type_z<<61 | (static_cast<inst_t>(1)<<target));
		}else if(gate[0] == 'H' && gate[1] == '\0'){
			std::size_t target;
			std::scanf("%lu", &target);
			std::cout<<gate<<" "<<target<<std::endl;
			insts_vec.push_back(inst_type_h<<61 | (static_cast<inst_t>(1)<<target));
		}else if(gate[0] == 'C' && gate[1] == 'X' && gate[2] == '\0'){
			std::size_t target, ctrl;
			std::scanf("%lu%lu", &target, &ctrl);
			std::cout<<gate<<" "<<target<<" "<<ctrl<<std::endl;
			insts_vec.push_back(inst_type_cx<<61 | (static_cast<inst_t>(ctrl) << 32) | (static_cast<inst_t>(1)<<target));
		}else if(gate[0] == 'C' && gate[1] == 'Z' && gate[2] == '\0'){
			std::size_t target, ctrl;
			std::scanf("%lu%lu", &target, &ctrl);
			std::cout<<gate<<" "<<target<<" "<<ctrl<<std::endl;
			insts_vec.push_back(inst_type_cz<<61 | (static_cast<inst_t>(ctrl) << 32) | (static_cast<inst_t>(1)<<target));
		}else if(gate[0] == 'C' && gate[1] == 'C' && gate[2] == 'X' && gate[3] == '\0'){
			std::size_t target, ctrl_0, ctrl_1;
			std::scanf("%lu%lu%lu", &target, &ctrl_0, &ctrl_1);
			insts_vec.push_back(inst_type_ccx<<61 | (static_cast<inst_t>(ctrl_1) << 37) | (static_cast<inst_t>(ctrl_0) << 32) | (static_cast<inst_t>(1)<<target));
		}
	}
}
